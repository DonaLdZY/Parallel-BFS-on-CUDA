#include<iostream>
#include<fstream>
#include<vector>
#include<time.h>
#include<stdlib.h>
#include<random>
#include<string>
#include<string.h>
#include<queue>
#include <chrono>
#include<stdio.h>
#include<hip/hip_runtime.h>
using namespace std;
int n,m;
#define THREADS_PER_BLOCK 32

__global__ void bfs_kernel(int n, int m, int *d_index, int *d_edge, int *d_result, int h_cur_qsize, int * d_cur_q, int *d_nxt_qsize, int *d_nxt_q) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < h_cur_qsize) {
        int index=d_cur_q[tid];

        for(int i = d_index[index]; i < d_index[index + 1]; i++) {
            int y = d_edge[i];
            if(d_result[y] == -1 || d_result[y] > d_result[index] + 1) {
                d_result[y] = d_result[index] + 1;
                int nxt_qpos = atomicAdd(d_nxt_qsize, 1);
                d_nxt_q[nxt_qpos]=y;
            }
        }
    }
}

void bfs_task_parallel(int n, int m, int *index, int *edge, int *result) {
    const int n_blocks=(n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    int *d_index, *d_edge, *d_result;
    hipMalloc((void**)&d_index, (n+1)*sizeof(int));
    hipMalloc((void**)&d_edge, (m+1)*sizeof(int));
    hipMalloc((void**)&d_result, (n+1)*sizeof(int));
    hipMemcpy(d_index, index, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edge, edge, (m+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, (n+1)*sizeof(int), hipMemcpyHostToDevice);

    int *d_first_q, *d_second_q, *d_nxt_qsize;
    int h_cur_qsize=1;
    int zero=0;
    hipMalloc((void **)&d_first_q, (n+1)*sizeof(int));
	hipMalloc((void **)&d_second_q, (n+1)*sizeof(int));
	hipMalloc((void **)&d_nxt_qsize, sizeof(int));

    hipMemcpy(d_first_q, &zero, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_nxt_qsize, &zero, sizeof(int), hipMemcpyHostToDevice);
    
    auto startTime = chrono::steady_clock::now();
    int level=0;
    while (h_cur_qsize){
        int *d_cur_q, *d_nxt_q;
        if (level % 2==0){
            d_cur_q = d_first_q;
            d_nxt_q = d_second_q;
        }
        else{
            d_cur_q = d_second_q;
            d_nxt_q = d_first_q;
        }
            
        bfs_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(n, m, d_index, d_edge, d_result, h_cur_qsize, d_cur_q, d_nxt_qsize, d_nxt_q);
        hipDeviceSynchronize();
        hipMemcpy(&h_cur_qsize, d_nxt_qsize, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(d_nxt_qsize, &zero, sizeof(int), hipMemcpyHostToDevice);
        level++;
    }

    hipMemcpy(result, d_result, (n+1)*sizeof(int), hipMemcpyDeviceToHost);
    auto endTime = std::chrono::steady_clock::now();
	auto duration = chrono::duration_cast<chrono::milliseconds>(endTime - startTime).count();
	printf("Elapsed time for task_parallel BFS (without copying graph) : %li ms.\n", duration);
    hipFree(d_index);
    hipFree(d_edge);
    hipFree(d_result);
    hipFree(d_first_q);
    hipFree(d_second_q);
    hipFree(d_nxt_qsize);
}

int main() {
    std::cout<<"File name :"<<std::endl;
    string file;
    cin>>file;
    string file_in="data/"+file;
    string file_out="result/task_parallel_"+file;
    ifstream fin(file_in,ios::in);
    fin>>n>>m;
    int *index=new int[n+1]; //节点x的边的偏移量
    int *edge=new int[m+1]; //所有边
    index[0]=0;
    for (int i=0;i<n;i++){
        int xs;
        fin>>xs;
        index[i+1]=index[i]+xs;
        for (int j=0;j<xs;j++)
            fin>>edge[index[i]+j];
    }
    int *result=new int[n+1];
    for (int i=0;i<n;i++)
        result[i]=-1;
    result[0]=0;

    auto startTime = chrono::steady_clock::now();
    bfs_task_parallel(n,m,index,edge,result);
    auto endTime = std::chrono::steady_clock::now();
    long duration = chrono::duration_cast<chrono::milliseconds>(endTime - startTime).count();
	printf("Elapsed time for task_parallel BFS (with graph copying) : %li ms.\n", duration);

    ofstream fout(file_out,ios::out);
    fout<<n<<endl;
    for (int i=0;i<n;i++){
        fout<<result[i]<<endl;
    }
    delete []index;
    delete []edge;
    delete []result;
    fin.close();
    fout.close();
    return 0;
}

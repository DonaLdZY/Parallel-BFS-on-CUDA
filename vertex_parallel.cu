#include<iostream>
#include<fstream>
#include<vector>
#include<time.h>
#include<stdlib.h>
#include<random>
#include<string>
#include<string.h>
#include<queue>
#include <chrono>
#include<stdio.h>
#include<hip/hip_runtime.h>
using namespace std;
int n,m;
#define THREADS_PER_BLOCK 32

__global__ void bfs_kernel(int n, int m, int *d_index, int *d_edge, int *d_result, bool *d_continue, int level) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < n && d_result[index] == level) {
        for(int i = d_index[index]; i < d_index[index + 1]; i++) {
            int y = d_edge[i];
            if(d_result[y] == -1 || d_result[y] > d_result[index] + 1) {
                d_result[y] = d_result[index] + 1;
                *d_continue = true;
            }
        }
    }
}

void bfs_vertex_parallel(int n, int m, int *index, int *edge, int *result) {
    const int n_blocks=(n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    int *d_index, *d_edge, *d_result;
    bool h_continue, *d_continue;

    hipMalloc((void**)&d_index, (n+1)*sizeof(int));
    hipMalloc((void**)&d_edge, (m+1)*sizeof(int));
    hipMalloc((void**)&d_result, (n+1)*sizeof(int));
    hipMalloc((void**)&d_continue, sizeof(bool));

    hipMemcpy(d_index, index, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edge, edge, (m+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    auto startTime = chrono::steady_clock::now();
    int level=0;
    do {
        h_continue = false;
        hipMemcpy(d_continue, &h_continue, sizeof(bool), hipMemcpyHostToDevice);
        bfs_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(n, m, d_index, d_edge, d_result, d_continue,level);
        hipDeviceSynchronize();
        hipMemcpy(&h_continue, d_continue, sizeof(bool), hipMemcpyDeviceToHost);
        level++;
    } while(h_continue);

    hipMemcpy(result, d_result, (n+1)*sizeof(int), hipMemcpyDeviceToHost);
    auto endTime = std::chrono::steady_clock::now();
	auto duration = chrono::duration_cast<chrono::milliseconds>(endTime - startTime).count();
	printf("Elapsed time for vertex_parallel BFS (without copying graph) : %li ms.\n", duration);

    hipFree(d_index);
    hipFree(d_edge);
    hipFree(d_result);
    hipFree(d_continue);
}

int main() {
    std::cout<<"File name :"<<std::endl;
    string file;
    cin>>file;
    string file_in="data/"+file;
    string file_out="result/vertex_parallel_"+file;
    ifstream fin(file_in,ios::in);
    fin>>n>>m;
    int *index=new int[n+1]; //节点x的边的偏移量
    int *edge=new int[m+1]; //所有边
    index[0]=0;
    for (int i=0;i<n;i++){
        int xs;
        fin>>xs;
        index[i+1]=index[i]+xs;
        for (int j=0;j<xs;j++)
            fin>>edge[index[i]+j];
    }
    int *result=new int[n+1];
    for (int i=0;i<n;i++)
        result[i]=-1;
    result[0]=0;

    auto startTime = chrono::steady_clock::now();
    bfs_vertex_parallel(n,m,index,edge,result);
    auto endTime = std::chrono::steady_clock::now();
    long duration = chrono::duration_cast<chrono::milliseconds>(endTime - startTime).count();
	printf("Elapsed time for vertex_parallel BFS (with graph copying) : %li ms.\n", duration);

    ofstream fout(file_out,ios::out);
    fout<<n<<endl;
    for (int i=0;i<n;i++){
        fout<<result[i]<<endl;
    }
    delete []index;
    delete []edge;
    delete []result;
    fin.close();
    fout.close();
    return 0;
}
